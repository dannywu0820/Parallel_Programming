/*********************************************************
*  DESCRIPTION:                                          *
*    Serial Concurrent Wave Equation - C Version         *
*    This program implements the concurrent wave equation*
**********************************************************/

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>

#define MAXPOINTS 1000000
#define MINPOINTS 20
#define MAXSTEPS 1000000
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update(void);
void printfinal(void);

int nsteps, /*number of time steps*/
    tpoints, /*total points along string*/
    rcode; /*generic return code*/

float values[MAXPOINTS + 2], /*values at time t*/
      oldval[MAXPOINTS + 2], /*values at time (t-dt)*/
      newval[MAXPOINTS + 2]; /*values at time (t+dt)*/
float v[MAXPOINTS + 2], /*serial used for comparing answers with parallel*/
      o[MAXPOINTS + 2],
      n[MAXPOINTS + 2];

/*********************************************************
*  Check input values from parameters                    *
**********************************************************/
void check_param(void){
    char tchar[20];
	
    /*check number of points, number of iterations*/
    while((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)){
        printf("Enter number of points along vibrating string [%d-%d]: ");
        scanf("%s", tchar);
	tpoints = atoi(tchar);
	if((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)){
            printf("Invalid. Please enter value between %d and %d\n", MINPOINTS, MAXPOINTS);
	}
    }
    while((nsteps < 1) || (nsteps > MAXSTEPS)){
	printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
	scanf("%s", tchar);
	nsteps = atoi(tchar);
	if((nsteps < 1) || (nsteps > MAXSTEPS)){
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
	}
    }
	
    printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}

/*********************************************************
*  Initialize points on line                             *
**********************************************************/
void init_line(void){
    int i, j;
    float x, fac, k, tmp;
	
    /*Calculate initial values based on sine curve*/
    fac = 2.0 * PI;
    k = 0.0;
    tmp = tpoints - 1;
    for(j = 1; j <= tpoints; j++){
	x = k/tmp;
	values[j] = sin(fac * x);
        v[j] = values[j];
	k = k + 1.0;
    }
	
    /*Initialize old values array*/
    for(i = 1; i <= tpoints; i++){
	oldval[i] = values[i];
        o[i] = v[i];
    }
}

/*********************************************************
*  Calculate new values using wave equation              *
**********************************************************/
__global__ void do_math_kernel(float *val_d, float *old_d, float *new_d, int num_of_steps, int num_of_points){
    float dtime, c, dx, tau, sqtau;
	
    dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;

    __shared__ float val_ds[1024];
    __shared__ float old_ds[1024];
    __shared__ float new_ds[1024];
    int tx = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    val_ds[tx] = val_d[index];
    old_ds[tx] = old_d[index];

    for(int i = 1; i <= num_of_steps; i++){
        /*This part needs to access elements from global memory of GPU*/
        /*if(index == 0 || index == num_of_points - 1) new_d[index] = 0.0;
        else new_d[index] = (2.0 * val_d[index]) - old_d[index] + (sqtau * (-2.0) * val_d[index]);
        old_d[index] = val_d[index];
        val_d[index] = new_d[index];*/
        /*This part accesses elements from shared memory of GPU -> faster*/
        /*if(index == 0 || index == num_of_points - 1) new_ds[tx] = 0.0;
        else new_ds[tx] = (2.0 * val_ds[tx]) - old_ds[tx] + (sqtau * (-2.0) * val_ds[tx]);
        old_ds[tx] = val_ds[tx];
        val_ds[tx] = new_ds[tx];*/
        /*This part only takes values[2~tpoints-1] total tpoins-2 points from CPU to GPU in order to reduce branch overhead*/ 
        new_ds[tx] = (2.0 * val_ds[tx]) - old_ds[tx] + (sqtau * (-2.0) * val_ds[tx]);
        old_ds[tx] = val_ds[tx];
        val_ds[tx] = new_ds[tx];
        
    }
    __syncthreads();
    val_d[index] = val_ds[tx];
    
}

void do_math(int i){
    float dtime, c, dx, tau, sqtau;
	
    dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;
    newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau * (-2.0) * values[i]);
    //n[i] = (2.0 * v[i]) - o[i] + (sqtau * (-2.0) * v[i]);
}

/***********************************************************
*  Update all values along line a specified number of times*
***********************************************************/
void updateOnDevice(){
    //int size = tpoints * sizeof(float);
    int size = (tpoints - 2) * sizeof(float);
    float *val_d, *old_d, *new_d; //memory on device

    /*1.Allocate device memory and move initiail values[] and oldval[] to GPU*/
    hipMalloc(&val_d, size);
    //cudaMemcpy(val_d, values+1, size, cudaMemcpyHostToDevice);
    hipMemcpy(val_d, values+2, size, hipMemcpyHostToDevice);
    hipMalloc(&old_d, size);
    //cudaMemcpy(old_d, oldval+1, size, cudaMemcpyHostToDevice);
    hipMemcpy(old_d, oldval+2, size, hipMemcpyHostToDevice);
    hipMalloc(&new_d, size);
	
    /*2.Invoke kernel function, each thread calculates a value[] element*/
    int threads_per_block, blocks_per_grid = tpoints/1024 + 1;
    if(tpoints > 1024) threads_per_block = 1024;
    else threads_per_block = tpoints;
    dim3 dimBlock(threads_per_block); 
    dim3 dimGrid(blocks_per_grid);
    do_math_kernel<<<dimGrid,dimBlock>>>(val_d, old_d, new_d, nsteps, tpoints);

    /*3.Read final results from GPU to CPU*/
    //cudaMemcpy(values+1, val_d, size, cudaMemcpyDeviceToHost);
    hipMemcpy(values+2, val_d, size, hipMemcpyDeviceToHost);
    hipFree(val_d); hipFree(old_d); hipFree(new_d);
}

void update(){
    int i, j;
	
    /*Update values for each time step*/
    for(i = 1; i <= nsteps; i++){
        /*Update points along line for this time step*/
        for(j = 1; j <= tpoints; j++){
	    /*global endpoints*/
            if((j == 1) || (j == tpoints)){
                newval[j] = 0.0;
                //n[j] = 0.0;
            }
	    else do_math(j);
	}
        /*Update old values with new values*/
        for(j = 1; j <= tpoints; j++){
	    oldval[j] = values[j];
	    values[j] = newval[j];
            /*o[j] = v[j];
            v[j] = n[j];*/
        }
    }
	
}

/**********************************************************
*  Print final results                                    *
**********************************************************/
void printfinal(){
    int i;

    for(i = 1; i <= tpoints; i++){
	printf("%6.4f ", values[i]);
	if(i%10 == 0) printf("\n");
    }	
}

/**********************************************************
*  Check serial and parallel answers                      *
**********************************************************/
void check_answer(){
    int wrong = 0, num = 0;
    for(int i = 1; i <= tpoints; i++){
        if(values[i]!=v[i]){
            wrong = 1;
            num++;
        }
    }
    if(wrong == 0) printf("right\n");
    else printf("%d are wrong\n", num);
    /*In command line ./ cuda_wave.out > [filename] to pipe output to the file
      then use diff file1 file2 to see if there is any difference*/
}

/*********************************************************
*  Main Program                                          *
**********************************************************/
int main(int argc, char *argv[]){
    sscanf(argv[1], "%d", &tpoints);
    sscanf(argv[2], "%d", &nsteps);
	
    check_param();
	
    printf("Initializing points on the line...\n");
    init_line();
    //printfinal();
	
    printf("Updating all points for all time steps...\n");
    //update();
    updateOnDevice();
	
    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");
	
    return 0;
}	  
